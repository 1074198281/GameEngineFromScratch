#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_function_table_definition.h>
#include <optix_stubs.h>
#include <optix_stack_size.h>

#include <hip/hip_runtime.h>

#include <iomanip>
#include <iostream>
#include <array>
#include <random>

#include "OptixTest.hpp"
#include "AssetLoader.hpp"
#include "Image.hpp"

// help functions 
std::mt19937 generator;
std::uniform_real_distribution<float> distribution(0.0, 1.0);
#define RND distribution(generator)

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
inline void check_cuda(hipError_t result, char const *const func,
                const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result)
                  << " (" << hipGetErrorString(result) << ") "
                  << " at " << file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

#define checkOptiXErrors(val) check_optix((val), #val, __FILE__, __LINE__)
inline void check_optix( OptixResult res, const char* call, const char* file, unsigned int line )
{
    if( res != OPTIX_SUCCESS )
    {
        std::cerr << "Optix call '" << call << "' failed: " << file << ':' << line << ")\n";
        exit(98);
    }
}

#define checkOptiXErrorsLog(val)                                                        \
    do {                                                                                \
        char    LOG[2048];                                                              \
        size_t  LOG_SIZE = sizeof(LOG);                                                 \
        check_optix_log((val), LOG, sizeof(LOG), LOG_SIZE, #val, __FILE__, __LINE__);   \
    } while (false)
inline void check_optix_log( OptixResult  res,
                           const char*  log,
                           size_t       sizeof_log,
                           size_t       sizeof_log_returned,
                           const char*  call,
                           const char*  file,
                           unsigned int line )
{
    if( res != OPTIX_SUCCESS )
    {
        std::cerr << "Optix call '" << call << "' failed: " << file << ':' << line << ")\nLog:\n"
           << log << ( sizeof_log_returned > sizeof_log ? "<TRUNCATED>" : "" ) << '\n';
    }
}

static void context_log_cb( unsigned int level, const char* tag, const char* message, void* /*cbdata */)
{
    std::cerr << "[" << std::setw( 2 ) << level << "][" << std::setw( 12 ) << tag << "]: "
    << message << "\n";
}

__global__ void rand_init(hiprandStateMRG32k3a *rand_state, const unsigned int max_x, const unsigned int max_y) {
    // Each thread in a block gets unique seed
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    unsigned int pixel_index = j * max_x + i;
    hiprand_init(2023 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

int main() {
    // Initialize CUDA and create OptiX context
    OptixDeviceContext context = nullptr;
    {
        checkCudaErrors(hipFree(0));

        hipCtx_t cuCtx = 0;

        checkOptiXErrors(optixInit());
        OptixDeviceContextOptions options = {};
        options.logCallbackFunction       = &context_log_cb;
        options.logCallbackLevel          = 4;
        checkOptiXErrors(optixDeviceContextCreate(cuCtx, &options, &context));
    }

    // accel handling
    constexpr int scene_obj_num = 22 * 22 + 1 + 3;
    OptixTraversableHandle  gas_handle;
    hipDeviceptr_t             d_gas_output_buffer;
    {
        std::array<float3, scene_obj_num>  sphereVertex;
        std::array<float,  scene_obj_num>  sphereRadius;
        std::array<uint16_t, scene_obj_num> g_mat_indices;

        int index = 0;
        // ground
        g_mat_indices[index]    = index; 
        sphereVertex[index]     = make_float3(0, -1000, -1);
        sphereRadius[index++]   = 1000.f;

        // random spheres
        for (int a = -11; a < 11; a++) {
            for (int b = -11; b < 11; b++) {
                g_mat_indices[index]    = index; 
                sphereVertex[index]     = make_float3(a + RND, 0.2f, b + RND);
                sphereRadius[index++]   = 0.2f;
            }
        }

        // three big balls
        g_mat_indices[index]    = index; 
        sphereVertex[index]     = make_float3(0, 1, 0);
        sphereRadius[index++]   = 1.0f;

        g_mat_indices[index]    = index; 
        sphereVertex[index]     = make_float3(-4, 1, 0);
        sphereRadius[index++]   = 1.0f;

        g_mat_indices[index]    = index; 
        sphereVertex[index]     = make_float3(4, 1, 0);
        sphereRadius[index++]   = 1.0f;

        assert(index == scene_obj_num);

        size_t sphereVertexSize = sizeof(sphereVertex[0]) * sphereVertex.size();
        size_t sphereRadiusSize = sizeof(sphereRadius[0]) * sphereRadius.size();

        hipDeviceptr_t d_vertex_buffer;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_vertex_buffer), sphereVertexSize));
        checkCudaErrors(hipMemcpy(reinterpret_cast<void *>(d_vertex_buffer), sphereVertex.data(), sphereVertexSize, hipMemcpyHostToDevice));

        hipDeviceptr_t d_radius_buffer;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_radius_buffer), sphereRadiusSize));
        checkCudaErrors(hipMemcpy(reinterpret_cast<void *>(d_radius_buffer), sphereRadius.data(), sphereRadiusSize, hipMemcpyHostToDevice));

        hipDeviceptr_t d_mat_indices;
        const size_t mat_indices_size_in_bytes = g_mat_indices.size() * sizeof(uint16_t);
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_mat_indices), mat_indices_size_in_bytes));
        checkCudaErrors(hipMemcpy(
            reinterpret_cast<void *>(d_mat_indices),
            g_mat_indices.data(),
            mat_indices_size_in_bytes,
            hipMemcpyHostToDevice
        ));

        std::array<uint32_t, scene_obj_num> sphere_input_flags;
        sphere_input_flags.fill(OPTIX_GEOMETRY_FLAG_DISABLE_ANYHIT);

        OptixBuildInput sphere_input = {};

        sphere_input.type                       = OPTIX_BUILD_INPUT_TYPE_SPHERES;
        sphere_input.sphereArray.vertexBuffers  = &d_vertex_buffer;
        sphere_input.sphereArray.numVertices    = sphereVertex.size();
        sphere_input.sphereArray.radiusBuffers  = &d_radius_buffer;

        sphere_input.sphereArray.flags          = sphere_input_flags.data();
        sphere_input.sphereArray.numSbtRecords  = scene_obj_num;
        sphere_input.sphereArray.sbtIndexOffsetBuffer       = d_mat_indices;
        sphere_input.sphereArray.sbtIndexOffsetSizeInBytes  = sizeof(uint16_t);
        sphere_input.sphereArray.sbtIndexOffsetStrideInBytes= sizeof(uint16_t);

        OptixAccelBuildOptions  accel_options = {};
        accel_options.buildFlags    =   OPTIX_BUILD_FLAG_ALLOW_COMPACTION | OPTIX_BUILD_FLAG_ALLOW_RANDOM_VERTEX_ACCESS;
        accel_options.operation     =   OPTIX_BUILD_OPERATION_BUILD;

        OptixAccelBufferSizes   gas_buffer_sizes;
        checkOptiXErrors(optixAccelComputeMemoryUsage(context, &accel_options, &sphere_input, 1, &gas_buffer_sizes));
        hipDeviceptr_t d_temp_buffer_gas;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_temp_buffer_gas), gas_buffer_sizes.tempSizeInBytes));

        // non-compacted output
        hipDeviceptr_t d_buffer_temp_output_gas_and_compacted_size;
        size_t      compactedSizeOffset = My::roundUp<size_t>(gas_buffer_sizes.outputSizeInBytes, 8ull);
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_buffer_temp_output_gas_and_compacted_size), compactedSizeOffset + 8));

        OptixAccelEmitDesc emitProperty = {};
        emitProperty.type               = OPTIX_PROPERTY_TYPE_COMPACTED_SIZE;
        emitProperty.result             = (hipDeviceptr_t)((char*)d_buffer_temp_output_gas_and_compacted_size + compactedSizeOffset);

        checkOptiXErrors(optixAccelBuild(context,
                                        0, // CUDA stream
                                        &accel_options,
                                        &sphere_input,
                                        1, // num build inputs
                                        d_temp_buffer_gas, gas_buffer_sizes.tempSizeInBytes,
                                        d_buffer_temp_output_gas_and_compacted_size, gas_buffer_sizes.outputSizeInBytes,
                                        &gas_handle,
                                        &emitProperty,
                                        1));

        d_gas_output_buffer = d_buffer_temp_output_gas_and_compacted_size;

        checkCudaErrors(hipFree((void *)d_temp_buffer_gas));
        checkCudaErrors(hipFree((void *)d_mat_indices));
        checkCudaErrors(hipFree((void *)d_vertex_buffer));
        checkCudaErrors(hipFree((void *)d_radius_buffer));

        size_t compacted_gas_size;
        checkCudaErrors(hipMemcpy(&compacted_gas_size, (void *)emitProperty.result, sizeof(size_t), hipMemcpyDeviceToHost));

        if(compacted_gas_size < gas_buffer_sizes.outputSizeInBytes) {
            checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_gas_output_buffer), compacted_gas_size));

            // use handle as input and output
            checkOptiXErrors(optixAccelCompact(context, 0, gas_handle, d_gas_output_buffer, compacted_gas_size, &gas_handle));

            checkCudaErrors(hipFree((void *)d_buffer_temp_output_gas_and_compacted_size));
        } else {
            d_gas_output_buffer = d_buffer_temp_output_gas_and_compacted_size;
        }
    }
    
    // Create module
    OptixModule module = nullptr;
    OptixModule sphere_module = nullptr;
    OptixPipelineCompileOptions pipeline_compile_options = {};
    {
        OptixModuleCompileOptions module_compile_options = {};
#if !defined(NDEBUG)
        module_compile_options.optLevel     = OPTIX_COMPILE_OPTIMIZATION_LEVEL_0;
        module_compile_options.debugLevel   = OPTIX_COMPILE_DEBUG_LEVEL_FULL;
#endif
        pipeline_compile_options.usesMotionBlur         = false;
        pipeline_compile_options.traversableGraphFlags  = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;
        pipeline_compile_options.numPayloadValues       = 11;
        pipeline_compile_options.numAttributeValues     = 2;
#ifdef DEBUG
        pipeline_compile_options.exceptionFlags         = OPTIX_EXCEPTION_FLAG_DEBUG | OPTIX_EXCEPTION_FLAG_TRACE_DEPTH | OPTIX_EXCEPTION_FLAG_STACK_OVERFLOW;
#else
        pipeline_compile_options.exceptionFlags         = OPTIX_EXCEPTION_FLAG_NONE;
#endif
        pipeline_compile_options.pipelineLaunchParamsVariableName = "params";
        pipeline_compile_options.usesPrimitiveTypeFlags = OPTIX_PRIMITIVE_TYPE_FLAGS_SPHERE;

        My::AssetLoader assetLoader;
        auto shader = assetLoader.SyncOpenAndReadBinary("Shaders/CUDA/OptixTest.shader.optixir");

        checkOptiXErrorsLog(optixModuleCreateFromPTX(
            context,
            &module_compile_options,
            &pipeline_compile_options,
            (const char*)shader.GetData(),
            shader.GetDataSize(),
            LOG, &LOG_SIZE,
            &module
        ));

        OptixBuiltinISOptions builtin_is_options = {};

        builtin_is_options.usesMotionBlur       = false;
        builtin_is_options.builtinISModuleType  = OPTIX_PRIMITIVE_TYPE_SPHERE;
        checkOptiXErrors(optixBuiltinISModuleGet(context, &module_compile_options, &pipeline_compile_options,
                                                &builtin_is_options, &sphere_module));
    }

    // Create program groups
    OptixProgramGroup raygen_prog_group         = nullptr;
    OptixProgramGroup miss_prog_group           = nullptr;
    OptixProgramGroup hitgroup_prog_group       = nullptr;
    {
        OptixProgramGroupOptions program_group_options  = {};

        // ray gen group
        OptixProgramGroupDesc    raygen_prog_group_desc = {};
        raygen_prog_group_desc.kind                     = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
        raygen_prog_group_desc.raygen.module            = module;
        raygen_prog_group_desc.raygen.entryFunctionName = "__raygen__rg";
        checkOptiXErrorsLog(optixProgramGroupCreate(
            context,
            &raygen_prog_group_desc,
            1, // num program groups
            &program_group_options,
            LOG, &LOG_SIZE,
            &raygen_prog_group
        ));

        // miss group
        OptixProgramGroupDesc miss_prog_group_desc  = {};
        miss_prog_group_desc.kind                   = OPTIX_PROGRAM_GROUP_KIND_MISS;
        miss_prog_group_desc.miss.module            = module;
        miss_prog_group_desc.miss.entryFunctionName = "__miss__ms";

        checkOptiXErrorsLog(optixProgramGroupCreate(
            context,
            &miss_prog_group_desc,
            1,
            &program_group_options,
            LOG, &LOG_SIZE,
            &miss_prog_group
        )); 

        // hit group
        OptixProgramGroupDesc hitgroup_prog_group_desc          = {};
        hitgroup_prog_group_desc.kind                           = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
        hitgroup_prog_group_desc.hitgroup.moduleCH              = module;
        hitgroup_prog_group_desc.hitgroup.entryFunctionNameCH   = "__closesthit__ch";
        hitgroup_prog_group_desc.hitgroup.moduleAH              = nullptr;
        hitgroup_prog_group_desc.hitgroup.moduleIS              = sphere_module;
        hitgroup_prog_group_desc.hitgroup.entryFunctionNameIS   = nullptr;

        checkOptiXErrorsLog(optixProgramGroupCreate(
            context,
            &hitgroup_prog_group_desc,
            1,
            &program_group_options,
            LOG, &LOG_SIZE,
            &hitgroup_prog_group
        ));
    }

    // Link pipeline
    OptixPipeline pipeline = nullptr;
    {
        const uint32_t      max_trace_depth = 2;
        OptixProgramGroup   program_groups[] = { raygen_prog_group, miss_prog_group, hitgroup_prog_group };

        OptixPipelineLinkOptions pipeline_link_options = {};
        pipeline_link_options.maxTraceDepth     = max_trace_depth;
        pipeline_link_options.debugLevel        = OPTIX_COMPILE_DEBUG_LEVEL_FULL;

        checkOptiXErrorsLog(optixPipelineCreate(
            context,
            &pipeline_compile_options,
            &pipeline_link_options,
            program_groups,
            sizeof(program_groups) / sizeof(program_groups[0]),
            LOG, &LOG_SIZE,
            &pipeline
        ) );

        OptixStackSizes stack_sizes = {};
        for (auto& prog_group : program_groups) {
            checkOptiXErrors(optixUtilAccumulateStackSizes(prog_group, &stack_sizes));
        }

        uint32_t direct_callable_stack_size_from_traversal;
        uint32_t direct_callable_stack_size_from_state;
        uint32_t continuation_stack_size;
        checkOptiXErrors(optixUtilComputeStackSizes(&stack_sizes, max_trace_depth,
                                                    0, // maxCCDepth
                                                    0, // maxDCDepth
                                                    &direct_callable_stack_size_from_traversal,
                                                    &direct_callable_stack_size_from_state,
                                                    &continuation_stack_size));

        const uint32_t max_traversal_depth = 1;
        checkOptiXErrors(optixPipelineSetStackSize(pipeline, direct_callable_stack_size_from_traversal,
                                                    direct_callable_stack_size_from_state,
                                                    continuation_stack_size,
                                                    max_traversal_depth ));
    }

    // Set up shader binding table
    OptixShaderBindingTable sbt = {};
    {
        // ray gen SBT
        hipDeviceptr_t     d_raygen_record;
        const size_t    raygen_record_size = sizeof(RayGenSbtRecord);
        checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_raygen_record), raygen_record_size));

        RayGenSbtRecord rg_sbt = {};
        checkOptiXErrors(optixSbtRecordPackHeader(raygen_prog_group, &rg_sbt));
        checkCudaErrors(hipMemcpy(
            reinterpret_cast<void*>(d_raygen_record),
            &rg_sbt,
            raygen_record_size,
            hipMemcpyHostToDevice
        ));

        // miss SBT
        hipDeviceptr_t d_miss_record;
        size_t      miss_record_size = sizeof(MissSbtRecord);
        checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_miss_record), miss_record_size));

        MissSbtRecord ms_sbt = {};
        ms_sbt.data.bg_color = {0.5f, 0.7f, 1.0f};
        checkOptiXErrors(optixSbtRecordPackHeader(miss_prog_group, &ms_sbt));
        checkCudaErrors(hipMemcpy(
            reinterpret_cast<void**>(d_miss_record),
            &ms_sbt,
            miss_record_size,
            hipMemcpyHostToDevice
        ));

        // Hit SBT
        constexpr int material_count = scene_obj_num;
        hipDeviceptr_t d_hitgroup_record;
        size_t      hitgroup_record_size = sizeof(HitGroupSbtRecord);
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_hitgroup_record), hitgroup_record_size * material_count));
        std::array<HitGroupSbtRecord, material_count> hg_sbt;

        int index = 0;
        // ground
        checkOptiXErrors(
            optixSbtRecordPackHeader(hitgroup_prog_group, &hg_sbt[index]));
        hg_sbt[index].data.material_type = Material::MAT_DIFFUSE;
        hg_sbt[index++].data.base_color = {0.5f, 0.5f, 0.5f};

        // random spheres
        for (int a = -11; a < 11; a++) {
            for (int b = -11; b < 11; b++) {
                float choose_mat = RND;
                if (choose_mat < 0.8f) {
					checkOptiXErrors(
						optixSbtRecordPackHeader(hitgroup_prog_group, &hg_sbt[index]));
                    hg_sbt[index].data.material_type = Material::MAT_DIFFUSE;
                    hg_sbt[index++].data.base_color = {RND * RND, RND * RND, RND * RND};
                } else if (choose_mat < 0.95f) {
					checkOptiXErrors(
						optixSbtRecordPackHeader(hitgroup_prog_group, &hg_sbt[index]));
                    hg_sbt[index].data.material_type = Material::MAT_METAL;
                    hg_sbt[index].data.base_color = {0.5f * (1.0f + RND), 0.5f * (1.0f + RND), 0.5f * (1.0f + RND)};
                    hg_sbt[index++].data.fuzz = 0.5f * (1.0f + RND);
                } else {
					checkOptiXErrors(
						optixSbtRecordPackHeader(hitgroup_prog_group, &hg_sbt[index]));
                    hg_sbt[index].data.material_type = Material::MAT_DIELECTRIC;
                    hg_sbt[index++].data.ir = 1.5f;
                }
            }
        }

		checkOptiXErrors(
			optixSbtRecordPackHeader(hitgroup_prog_group, &hg_sbt[index]));
        hg_sbt[index].data.material_type = Material::MAT_DIELECTRIC;
        hg_sbt[index++].data.base_color = 1.5f;

		checkOptiXErrors(
			optixSbtRecordPackHeader(hitgroup_prog_group, &hg_sbt[index]));
        hg_sbt[index].data.material_type = Material::MAT_DIFFUSE;
        hg_sbt[index++].data.base_color = {0.4f, 0.2f, 0.1f};

		checkOptiXErrors(
			optixSbtRecordPackHeader(hitgroup_prog_group, &hg_sbt[index]));
        hg_sbt[index].data.material_type = Material::MAT_METAL;
        hg_sbt[index].data.base_color = {0.7f, 0.6f, 0.5f};
        hg_sbt[index++].data.fuzz = 0.0f;

        assert(index == material_count);

        checkCudaErrors(hipMemcpy(
            reinterpret_cast<void *>(d_hitgroup_record),
            hg_sbt.data(),
            hitgroup_record_size * material_count,
            hipMemcpyHostToDevice
        ));

        sbt.raygenRecord                = d_raygen_record;
        sbt.missRecordBase              = d_miss_record;
        sbt.missRecordStrideInBytes     = sizeof(MissSbtRecord);
        sbt.missRecordCount             = 1;
        sbt.hitgroupRecordBase          = d_hitgroup_record;
        sbt.hitgroupRecordStrideInBytes = sizeof(HitGroupSbtRecord);
        sbt.hitgroupRecordCount         = material_count;
    }

    // Render Settings
    My::Image img;
    My::Image* d_img;
    My::RayTracingCamera<float>* d_camera;
    hiprandStateMRG32k3a* d_rand_state;
    {
        const float aspect_ratio = 16.0 / 9.0;
        const int image_width = 1920;
        const int image_height = static_cast<int>(image_width / aspect_ratio);

        // Canvas
        img.Width = image_width;
        img.Height = image_height;
        img.bitcount = 96; 
        img.bitdepth = 32;
        img.pixel_format = My::PIXEL_FORMAT::RGB32;
        img.pitch = (img.bitcount >> 3) * img.Width;
        img.compressed = false;
        img.compress_format = My::COMPRESSED_FORMAT::NONE;
        img.data_size = img.Width * img.Height * (img.bitcount >> 3);
        auto num_pixels = image_width * image_height;

        checkCudaErrors(hipMallocManaged((void **)&img.data, img.data_size));

        checkCudaErrors(hipMalloc((void **)&d_img, sizeof(My::Image)));
        checkCudaErrors(hipMemcpy((void *)d_img, &img, sizeof(My::Image), hipMemcpyHostToDevice));

        point3 lookfrom{13, 2, 3};
        point3 lookat{0, 0, 0};
        vec3 vup{0, 1, 0};
        auto dist_to_focus = 10.0f;
        auto aperture = 0.1f;

        My::RayTracingCamera<float> camera (lookfrom, lookat, vup, 20.0f, aspect_ratio,
                                aperture, dist_to_focus);

        checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(My::RayTracingCamera<float>)));
        checkCudaErrors(hipMemcpy((void **)d_camera, &camera, sizeof(My::RayTracingCamera<float>), hipMemcpyHostToDevice)); 

        int tile_width = 8;
        int tile_height = 8;

        dim3 blocks((image_width + tile_width - 1) / tile_width, (image_height + tile_height - 1) / tile_height);
        dim3 threads(tile_width, tile_height);

        checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels * sizeof(hiprandStateMRG32k3a)));

        rand_init<<<blocks, threads>>>(d_rand_state, image_width, image_height);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
    }
    // launch
    {
        hipStream_t stream;
        checkCudaErrors(hipStreamCreate(&stream));

        Params params;
        params.handle       = gas_handle;
        params.image        = d_img;
        params.cam          = d_camera;
        params.rand_state   = d_rand_state;
        params.max_depth    = 50;
        params.num_of_samples = 512;

        hipDeviceptr_t d_param;
        checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_param), sizeof(Params)));
        checkCudaErrors(hipMemcpy(
            reinterpret_cast<void**>(d_param),
            &params, sizeof(params),
            hipMemcpyHostToDevice
        ));

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        checkOptiXErrors(optixLaunch(pipeline, stream, d_param, sizeof(Params), &sbt, img.Width, img.Height, 1));
        hipEventRecord(stop);

        checkCudaErrors(hipDeviceSynchronize());

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Rendering time: %f ms\n", milliseconds);

        img.SaveTGA("raytracing_optix.tga");
        img.data = nullptr;  // to avoid double free

        // clean up
        {
            checkCudaErrors(hipFree(reinterpret_cast<void*>(d_rand_state)));
            checkCudaErrors(hipFree(reinterpret_cast<void*>(d_img)));
            checkCudaErrors(hipFree(reinterpret_cast<void*>(d_camera)));
            checkCudaErrors(hipFree(reinterpret_cast<void*>(d_param)));

            checkCudaErrors(hipFree(reinterpret_cast<void*>(sbt.raygenRecord)));
            checkCudaErrors(hipFree(reinterpret_cast<void*>(sbt.missRecordBase)));
            checkCudaErrors(hipFree(reinterpret_cast<void*>(sbt.hitgroupRecordBase)));
            checkCudaErrors(hipFree(reinterpret_cast<void*>(d_gas_output_buffer)));

            checkOptiXErrors(optixPipelineDestroy(pipeline));
            checkOptiXErrors(optixProgramGroupDestroy(hitgroup_prog_group));
            checkOptiXErrors(optixProgramGroupDestroy(miss_prog_group));
            checkOptiXErrors(optixProgramGroupDestroy(raygen_prog_group));
            checkOptiXErrors(optixModuleDestroy(module));
            checkOptiXErrors(optixModuleDestroy(sphere_module));

            checkOptiXErrors(optixDeviceContextDestroy(context));
        }
    }

    return 0;
}