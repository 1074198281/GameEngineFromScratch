#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include "geommath.hpp"
#include "BVH.hpp"
#include "Sphere.hpp"

#include "TestMaterial.hpp"

using color = My::Vector3<float>;
using point3 = My::Point<float>;
using vec3 = My::Vector3<float>;

// help functions 
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func,
                const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result)
                  << " (" << hipGetErrorString(result) << ") "
                  << " at " << file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(2023, 0, 0, rand_state);
    }
}

__global__ void test(hiprandState *local_rand_state) {
    const int scene_obj_num = 1;
    My::Hitable<float>** pList = new My::Hitable<float>*[scene_obj_num];
    for (int i = 0; i < scene_obj_num; i++) {
        pList[i] = new My::Sphere<float, material *>(1000.0f, point3({0, -1000, -1}),
                              new lambertian(vec3({0.5, 0.5, 0.5})));
    }

    My::SimpleBVHNode<float>* pWorld = new My::SimpleBVHNode<float>(pList, 0, scene_obj_num, local_rand_state);
    delete pList;
    delete pWorld;
}

int main() {
    hiprandState *d_rand_state_1;

    checkCudaErrors(hipMalloc((void **)&d_rand_state_1, sizeof(hiprandState)));

    rand_init<<<1, 1>>>(d_rand_state_1);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    test<<<1, 1>>>(d_rand_state_1);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipFree(d_rand_state_1));

    return 0;
}